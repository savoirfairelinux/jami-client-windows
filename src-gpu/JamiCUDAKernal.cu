#include "hip/hip_runtime.h"
// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <cstdint>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include ""


__global__ void
seperateUVChannelNV12_Kernal(uint8_t* UVbuf, uint8_t* Ubuf, uint8_t* Vbuf, int linesize, int widthOfOutPut, int height)
{
    // the thread index should be the same as the output array's index ,which is twice of the input's index
    const unsigned int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if(tidX < widthOfOutPut && tidY < height) {
        unsigned int indexUInUVArray = linesize * tidY + (2 * tidX - 1);
        unsigned int indexVInUVArray = linesize * tidY + (2 * tidX);

        unsigned int indexInOutputArray = widthOfOutPut * tidY + tidX;
        Ubuf[indexInOutputArray] = UVbuf[indexUInUVArray];
        Vbuf[indexInOutputArray] = UVbuf[indexVInUVArray];
    }
}

extern "C" bool
seperateUVChannelNV12()
{
    bool result = false;
}