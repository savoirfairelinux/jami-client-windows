#include "hip/hip_runtime.h"
// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <cstdint>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include ""


__global__ void
    seperateUVChannelNV12_Kernal(uint8_t* UVbuf,int linesize, int width, int height , uint8_t* Ubuf, uint8_t* Vbuf)
{
    // the thread index should be the same as the output array's index
    const unsigned int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int tidY = blockDim.y * blockIdx.y + threadIdx.y;


}

extern "C" void
    seperateUVChannelNV12()
{

}