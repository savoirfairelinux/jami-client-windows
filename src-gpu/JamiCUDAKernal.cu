#include "hip/hip_runtime.h"
// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <cstdint>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include ""


__global__ void
seperateUVChannelNV12_Kernal(uint8_t* UVbuf, uint8_t* Ubuf, uint8_t* Vbuf, int linesize, int widthOfOutPut, int height)
{
    // the thread index should be the same as the output array's index ,which is twice of the input's index
    const unsigned int tidX = blockDim.x * blockIdx.x + threadIdx.x;
    const unsigned int tidY = blockDim.y * blockIdx.y + threadIdx.y;

    if(tidX < widthOfOutPut && tidY < height) {
        unsigned int indexUInUVArray = linesize * tidY + (2 * tidX - 1);
        unsigned int indexVInUVArray = linesize * tidY + (2 * tidX);

        unsigned int indexInOutputArray = widthOfOutPut * tidY + tidX;
        Ubuf[indexInOutputArray] = UVbuf[indexUInUVArray];
        Vbuf[indexInOutputArray] = UVbuf[indexVInUVArray];
    }
}

extern "C" void
seperateUVChannelNV12(uint8_t * UVbuf, uint8_t * Ubuf, uint8_t * Vbuf, int linesize, int widthOfOutPut, int height)
{
    unsigned int blockDimX = 32;
    unsigned int blockDimY = 4;

    unsigned int overallBlockNumX = widthOfOutPut % blockDimX == 0 ? widthOfOutPut / blockDimX : (widthOfOutPut / blockDimX) + 1;
    unsigned int overallBlockNumY = height % blockDimY == 0 ? height / blockDimY : (height / blockDimY) + 1;

    dim3 blockNumPerGrid(overallBlockNumX, overallBlockNumY,1);
    dim3 threadNumPerBlock(blockDimX, blockDimY,1);

    seperateUVChannelNV12_Kernal <<<blockNumPerGrid, threadNumPerBlock >>>(UVbuf, Ubuf, Vbuf, linesize, widthOfOutPut, height);
}